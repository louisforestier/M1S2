#include "chronoGPU.hpp"
#include <hip/hip_runtime_api.h>
#include <iostream>

ChronoGPU::ChronoGPU() 
	: m_started( false ) {
		checkCudaErrors( hipEventCreate( &m_start ) );
		checkCudaErrors( hipEventCreate( &m_end ) );
}

ChronoGPU::~ChronoGPU() {
	if ( m_started ) {
		stop();
		std::cerr << "ChronoGPU::~ChronoGPU(): hrono wasn't turned off!" << std::endl; 
	}
	checkCudaErrors( hipEventDestroy( m_start ) );
	checkCudaErrors( hipEventDestroy( m_end ) );
}

void ChronoGPU::start() {
	if ( !m_started ) {
		checkCudaErrors( hipEventRecord( m_start, 0 ) );
		m_started = true;
	}
	else
		std::cerr << "ChronoGPU::start(): chrono is already started!" << std::endl;
}

void ChronoGPU::stop() {
	if ( m_started ) {
		checkCudaErrors( hipEventRecord( m_end, 0 ) );
		checkCudaErrors( hipEventSynchronize( m_end ) );
		m_started = false;
	}
	else
		std::cerr << "ChronoGPU::stop(): chrono wasn't started!" << std::endl;
}

float ChronoGPU::elapsedTime() { 
	float time = 0.f;
	checkCudaErrors( hipEventElapsedTime( &time, m_start, m_end ) );
	return time;
}
