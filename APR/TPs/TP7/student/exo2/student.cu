#include "hip/hip_runtime.h"
#include <iostream>
#include <exo2/student.h>
#include <OPP_cuda.cuh>

using uchar = unsigned char;

namespace 
{
	// L'opération est associative (enfin, en toute généralité), et donc les permutations de valeurs sont interdites.
	// Seul les changements de parenthèses sont autorisées ...
	// Donc il y a deux solutions :
	// - La plus simple est d'effectuer plusieurs réductions successives par blocs
	// - La plus difficile mais efficace, et de grouper les valeurs consécutives par thread.
	// Avec cette seconde, le premier thread (0) va traiter des valeurs consécutives. Le thread suivant aussi, etc.
	// En supposant par exemple que chaque thread traite 4 valeurs, alors les 4 premiers pixels du blocs sont utilisés par
	// le thread 0, le 4 suivant par le thread 1, etc. jusqu'au thread 255 ;-)
	// NB : on suppose que le nombre de warps est une puissance de 2 (et donc divise 1024)
	template<int NB_WARPS>
	__device__ 
	__forceinline__
	void loadSharedMemoryAssociate(float const*const data) 
	{
		float*const shared = OPP::CUDA::getSharedMemory<float>();

		const auto globalOffset = 1024 * blockIdx.x;
		const auto localThreadId = threadIdx.x;
		const unsigned nbPixelsPerThread = (1024 + 32*NB_WARPS - 1) / (32*NB_WARPS);

		float sumPerThread = 0.f;

		for(unsigned i=0; i<nbPixelsPerThread; ++i) 
		{
			// indice du pixel à traiter
			const auto pixelIdInBlock = nbPixelsPerThread * localThreadId + i;
			
			// TODO
		}
		shared[localThreadId] = sumPerThread;
		__syncthreads();
	}


	// idem exo1, sauf test de débordement
	template<int NB_WARPS>
	__device__ 
	__forceinline__
	void reduceJumpingStep(const int jump)
	{
		// TODO 
	}


	// on ne changera ici que le nombre d'itérations (10 avant, ici moins)
	template<int NB_WARPS>
	__device__
	__forceinline__
	float reducePerBlock(
		float const*const source
	) {
		// TODO
	}	
	

	// ressemble beaucoup à l'exo1 ...
	template<int NB_WARPS>
	__device__
	__forceinline__
	void fillBlock(
		const float color, 
		float*const result
	) {
		// calcul de l'offset du bloc : la taille est 1024
		const auto offset = blockIdx.x * 1024;
		// TODO
	}


	// idem exo1 with templates
	template<int NB_WARPS>
	struct EvaluateWarpNumber {
		enum { res = 1 };
	};
	template<>
	struct EvaluateWarpNumber<1> {
		enum { res = 16 };
	};
	template<>
	struct EvaluateWarpNumber<2> {
		enum { res = 8 };
	};
	template<>
	struct EvaluateWarpNumber<4> {
		enum { res = 4 };
	};
	template<>
	struct EvaluateWarpNumber<8> {
		enum { res = 4 };
	};
	template<>
	struct EvaluateWarpNumber<16> {
		enum { res = 2 };
	};

	// idem exo1
	template<int NB_WARPS=32>
	__global__
	__launch_bounds__(32*NB_WARPS , EvaluateWarpNumber<NB_WARPS>::res)
	void blockEffectKernel( 
		float const*const source, 
		float *const result
	) {
		const float sumInBlock = reducePerBlock<NB_WARPS>(source);
		fillBlock<NB_WARPS>(sumInBlock, result);
	}
}


// idem exo1, sauf la taille d'un bloc de threads (en Y)
void StudentWorkImpl::run_blockEffect(
	OPP::CUDA::DeviceBuffer<float>& dev_source,
	OPP::CUDA::DeviceBuffer<float>& dev_result,
	const unsigned nbWarps
) {
	// Le nombre de warps est réduit ...
	const auto size = dev_source.getNbElements();
	// Le nombre de threads par bloc dépend du nombre de warps ;-)
	dim3 threads(32 * nbWarps); 
	// Attention : le nombre de blocs est calculer en considérant des traitements de 1024 pixels ! 
	dim3 blocks ((size + 1024-1) / 1024 );
	// le reste est classique
	const size_t sizeSharedMemory(threads.x*sizeof(float));
	switch(nbWarps) {
		case 1:
			::blockEffectKernel<1> <<<blocks, threads, sizeSharedMemory>>>(
				dev_source.getDevicePointer(),
				dev_result.getDevicePointer()
			);
			return;
		case 2:
			::blockEffectKernel<2> <<<blocks, threads, sizeSharedMemory>>>(
				dev_source.getDevicePointer(),
				dev_result.getDevicePointer()
			);
			return;
		case 4:
			::blockEffectKernel<4> <<<blocks, threads, sizeSharedMemory>>>(
				dev_source.getDevicePointer(),
				dev_result.getDevicePointer()
			);
			return;
		case 8:
			::blockEffectKernel<8> <<<blocks, threads, sizeSharedMemory>>>(
				dev_source.getDevicePointer(),
				dev_result.getDevicePointer()
			);
			return;
		case 16:
			::blockEffectKernel<16> <<<blocks, threads, sizeSharedMemory>>>(
				dev_source.getDevicePointer(),
				dev_result.getDevicePointer()
			);
			return;
		case 32:
			::blockEffectKernel<32><<<blocks, threads, sizeSharedMemory>>>(
				dev_source.getDevicePointer(),
				dev_result.getDevicePointer()
			);
			return;
		default:
			::blockEffectKernel<32><<<blocks, threads, sizeSharedMemory>>>(
				dev_source.getDevicePointer(),
				dev_result.getDevicePointer()
			);
			return;
	}

}