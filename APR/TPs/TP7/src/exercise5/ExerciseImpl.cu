#include "hip/hip_runtime.h"
#define _USE_MATH_DEFINES
#include <cmath> 
#include <iostream>
#include <cstdlib>
#include <random>
#include <ppm.h>
#include <chronoCPU.hpp>
#include <exercise1/ImageBlockEffect.h>
#include <exercise5/ExerciseImpl.h>
#include <exercise5/ExerciseRunner.h>

namespace{

};

// ==========================================================================================
void ExerciseImpl::usage( const char*const prg ) {
    #ifdef WIN32
    const char*last_slash = strrchr(prg, '\\');
    #else
    const char*last_slash = strrchr(prg, '/');
    #endif
    std::cout << "Usage: " << (last_slash==nullptr ? prg : last_slash+1) 
        << " [ -i=<input.ppm> ]  [ -w=nb_warps] " << std::endl 
        << "where -i  specifies the file name of the input image," << std::endl
        << "  and -w  specifies the number of warps ("<<nbWarps<<" per default)."
        << std::endl;
}

// ==========================================================================================
void ExerciseImpl::usageAndExit( const char*const prg, const int code ) {
    usage(prg);
    exit( code );
}

// ==========================================================================================
void ExerciseImpl::displayHelpIfNeeded(const int argc, const char**argv) 
{
    if( checkCmdLineFlag(argc, argv, "-h") || checkCmdLineFlag(argc, argv, "help") ) {
        usageAndExit(argv[0], EXIT_SUCCESS);
    }
}
ExerciseImpl& ExerciseImpl::parseCommandLine(const int argc, const char**argv) 
{
    displayHelpIfNeeded(argc, argv);
    std::cout << "argv[1] = " << argv[1] << std::endl;
    if( checkCmdLineFlag(argc, argv, "i") ) {
        if( !getCmdLineArgumentString(argc, argv, "i", &inputFileName) ) 
        {
            std::cerr << "unable to get the input file name" << std::endl;
            usageAndExit(argv[0], -1);  
        }
        
    }
    if (checkCmdLineFlag(argc, argv, "w") ) {
        const int value = getCmdLineArgumentInt(argc, argv, "w"); 
        if( value < 1 ||value > 32 || !isPowerOfTwo(value) )
        {
            std::cerr << "The number of warps must be a power of two in [1..32]" << std::endl;
                usageAndExit(argv[0], -1);  
            }
        nbWarps = value;
    }
    return *this;
}

bool ExerciseImpl::isPowerOfTwo(int value)
{
    int nb = 0;
    while(value) 
    {
        nb += (value & 1);
        value >>= 1;
    }
    return nb == 1;
}

void ExerciseImpl::prepare_data() 
{
    std::cout << "- load input file " << inputFileName << std::endl;
    sourceImage = new PPMBitmap(inputFileName);
    const unsigned width = sourceImage->getWidth();
    const unsigned height = sourceImage->getHeight();
    std::cout << "- File <" << inputFileName << "> loaded. Contains "<<width<< " per " << height << " pixels." << std::endl;
    destImage = new PPMBitmap(width, height);
    prepare_truth();
}


void ExerciseImpl::prepare_truth() 
{
    trustedImage = new PPMBitmap(sourceImage->getWidth(), sourceImage->getHeight());
    std::cout << "Calculate the reference on CPU ..." << std::endl;    
    const unsigned nbTry = 10u;
    execute_and_display_time(
        true, 
        [&]() { ImageBlockEffect(*sourceImage, *trustedImage).applyBlockEffet(); }, 
        nbTry,
        std::string("\tDone in")
    );
}


void ExerciseImpl::run(const bool verbose) {    
    prepare_data();
    const unsigned nbTry = 25u;
    if( verbose ) {
        std::cout << "Student code will run " << nbTry << " times for statistics using " << nbWarps << " warps ..." << std::endl;
    }

    ExerciseRunner runner(sourceImage, nbWarps);
	
    execute_and_display_GPU_time(verbose, [&]() {
        runner.run(reinterpret_cast<StudentWorkImpl*>(student));
    }, nbTry);

	runner.copyTo(destImage);
}


bool ExerciseImpl::check() 
{
    saveTo(makeFileName(inputFileName, "_block_reference.ppm"), *trustedImage);
    saveTo(makeFileName(inputFileName, "_block_student.ppm"), *destImage);
    return checkImagesAreEquals(*trustedImage, *destImage);
}

std::string ExerciseImpl::makeFileName(const char*fileName, const std::string& extension)
{
    std::string s_output(fileName);
    size_t delimiter_pos = s_output.find_last_of('.');
    s_output.resize(delimiter_pos);
    s_output.append(extension);
    return s_output;
}
    
void ExerciseImpl::saveTo(const std::string& fileName, PPMBitmap& image)
{
    std::cout << "Save result into " << fileName << std::endl;
    image.saveTo(fileName.c_str());
}

bool ExerciseImpl::checkImagesAreEquals(const PPMBitmap&imageA, const PPMBitmap&imageB)
{
    if( imageA.getWidth() != imageB.getWidth() || imageA.getHeight() != imageB.getHeight() )
        return false;
    const uchar* ptrA = imageA.getPtr();
    const uchar* ptrB = imageB.getPtr();
    for(unsigned i=imageA.getWidth()*imageA.getWidth(); i--;)
        if( std::max(ptrA[i],ptrB[i]) - std::min(ptrA[i],ptrB[i]) > 1 ) return false;
    return true;
}
    